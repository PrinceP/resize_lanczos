
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <iostream>

#define ROUND_UP(f) ((int) ((f) >= 0.0 ? (f) + 0.5F : (f) - 0.5F))
#define UINT8 unsigned char
#define INT32 int
#define INT_MAX 0x7fffffff

/* pixel types */
#define IMAGING_TYPE_UINT8 0
#define IMAGING_TYPE_INT32 1
#define IMAGING_TYPE_FLOAT32 2
#define IMAGING_TYPE_SPECIAL 3 /* check mode for details */

#define IMAGING_MODE_LENGTH 6+1 /* Band names ("1", "L", "P", "RGB", "RGBA", "CMYK", "YCbCr", "BGR;xy") */


/* standard transforms */
#define IMAGING_TRANSFORM_AFFINE 0
#define IMAGING_TRANSFORM_PERSPECTIVE 2
#define IMAGING_TRANSFORM_QUAD 3


/* standard filters */
#define IMAGING_TRANSFORM_NEAREST 0
#define IMAGING_TRANSFORM_BOX 4
#define IMAGING_TRANSFORM_BILINEAR 2
#define IMAGING_TRANSFORM_HAMMING 5
#define IMAGING_TRANSFORM_BICUBIC 3
#define IMAGING_TRANSFORM_LANCZOS 1


typedef void (*ResampleFunction)(unsigned char *pOut, unsigned char *pIn, int offset,
                               int ksize, int *bounds, 
                               double *prekk, 
                               int inpWd, int inpHt, int inpStride, 
                               int outWd, int outHt, int outStride, 
                               int imType, int channels);

struct filter {
    double (*filter)(double x);
    double support;
};

static inline double box_filter(double x)
{
    if (x >= -0.5 && x < 0.5)
        return 1.0;
    return 0.0;
}

static inline double bilinear_filter(double x)
{
    if (x < 0.0)
        x = -x;
    if (x < 1.0)
        return 1.0-x;
    return 0.0;
}

static inline double hamming_filter(double x)
{
    if (x < 0.0)
        x = -x;
    if (x == 0.0)
        return 1.0;
    if (x >= 1.0)
        return 0.0;
    x = x * M_PI;
    return sin(x) / x * (0.54f + 0.46f * cos(x));
}

static inline double bicubic_filter(double x)
{
    /* https://en.wikipedia.org/wiki/Bicubic_interpolation#Bicubic_convolution_algorithm */
#define a -0.5
    if (x < 0.0)
        x = -x;
    if (x < 1.0)
        return ((a + 2.0) * x - (a + 3.0)) * x*x + 1;
    if (x < 2.0)
        return (((x - 5) * x + 8) * x - 4) * a;
    return 0.0;
#undef a
}

static inline double sinc_filter(double x)
{
    if (x == 0.0)
        return 1.0;
    x = x * M_PI;
    return sin(x) / x;
}

static inline double lanczos_filter(double x)
{
    /* truncated sinc */
    if (-3.0 <= x && x < 3.0)
        return sinc_filter(x) * sinc_filter(x/3);
    return 0.0;
}

static struct filter BOX = { box_filter, 0.5 };
static struct filter BILINEAR = { bilinear_filter, 1.0 };
static struct filter HAMMING = { hamming_filter, 1.0 };
static struct filter BICUBIC = { bicubic_filter, 2.0 };
static struct filter LANCZOS = { lanczos_filter, 3.0 };


/* 8 bits for result. Filter can have negative areas.
   In one cases the sum of the coefficients will be negative,
   in the other it will be more than 1.0. That is why we need
   two extra bits for overflow and int type. */
#define PRECISION_BITS (32 - 8 - 2)


/* Handles values form -640 to 639. */
UINT8 _clip8_lookups[1280] = {
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15,
    16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31,
    32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47,
    48, 49, 50, 51, 52, 53, 54, 55, 56, 57, 58, 59, 60, 61, 62, 63,
    64, 65, 66, 67, 68, 69, 70, 71, 72, 73, 74, 75, 76, 77, 78, 79,
    80, 81, 82, 83, 84, 85, 86, 87, 88, 89, 90, 91, 92, 93, 94, 95,
    96, 97, 98, 99, 100, 101, 102, 103, 104, 105, 106, 107, 108, 109, 110, 111,
    112, 113, 114, 115, 116, 117, 118, 119, 120, 121, 122, 123, 124, 125, 126, 127,
    128, 129, 130, 131, 132, 133, 134, 135, 136, 137, 138, 139, 140, 141, 142, 143,
    144, 145, 146, 147, 148, 149, 150, 151, 152, 153, 154, 155, 156, 157, 158, 159,
    160, 161, 162, 163, 164, 165, 166, 167, 168, 169, 170, 171, 172, 173, 174, 175,
    176, 177, 178, 179, 180, 181, 182, 183, 184, 185, 186, 187, 188, 189, 190, 191,
    192, 193, 194, 195, 196, 197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207,
    208, 209, 210, 211, 212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223,
    224, 225, 226, 227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239,
    240, 241, 242, 243, 244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
};

UINT8 *clip8_lookups = &_clip8_lookups[640];

static inline UINT8 clip8(int in)
{
    //printf("%d\n", in);
    return clip8_lookups[in >> PRECISION_BITS];
}


__device__ UINT8 _clip8_lookups_cuda[1280] = {
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15,
    16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31,
    32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47,
    48, 49, 50, 51, 52, 53, 54, 55, 56, 57, 58, 59, 60, 61, 62, 63,
    64, 65, 66, 67, 68, 69, 70, 71, 72, 73, 74, 75, 76, 77, 78, 79,
    80, 81, 82, 83, 84, 85, 86, 87, 88, 89, 90, 91, 92, 93, 94, 95,
    96, 97, 98, 99, 100, 101, 102, 103, 104, 105, 106, 107, 108, 109, 110, 111,
    112, 113, 114, 115, 116, 117, 118, 119, 120, 121, 122, 123, 124, 125, 126, 127,
    128, 129, 130, 131, 132, 133, 134, 135, 136, 137, 138, 139, 140, 141, 142, 143,
    144, 145, 146, 147, 148, 149, 150, 151, 152, 153, 154, 155, 156, 157, 158, 159,
    160, 161, 162, 163, 164, 165, 166, 167, 168, 169, 170, 171, 172, 173, 174, 175,
    176, 177, 178, 179, 180, 181, 182, 183, 184, 185, 186, 187, 188, 189, 190, 191,
    192, 193, 194, 195, 196, 197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207,
    208, 209, 210, 211, 212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223,
    224, 225, 226, 227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239,
    240, 241, 242, 243, 244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
};

__device__ UINT8 *clip8_lookups_cuda = &_clip8_lookups_cuda[640];

__device__ static inline UINT8 clip8_cuda(int in)
{
    //printf("%d\n", in);
    return clip8_lookups_cuda[in >> PRECISION_BITS];
}



int
precompute_coeffs(int inSize, float in0, float in1, int outSize,
                  struct filter *filterp, int **boundsp, double **kkp) {
    double support, scale, filterscale;
    double center, ww, ss;
    int xx, x, ksize, xmin, xmax;
    int *bounds;
    double *kk, *k;
    
    printf("precompute_coeffs\n");
    /* prepare for horizontal stretch */
    printf("outsize = %d :: in1 = %f :: in0 = %f \n", outSize, in1, in0);
    filterscale = scale = (double) (in1 - in0) / outSize;
    if (filterscale < 1.0) {
        filterscale = 1.0;
    }

    /* determine support size (length of resampling filter) */
    support = filterp->support * filterscale;

    /* maximum number of coeffs */
    ksize = (int) ceil(support) * 2 + 1;

    printf("ksize = %d\n", ksize);
    printf("support = %f\n", ceil(support));
    printf("filterscale = %f\n", filterscale);
    // check for overflow
    if (outSize > INT_MAX / (ksize * sizeof(double))) {
        return 0;
    }

    /* coefficient buffer */
    /* malloc check ok, overflow checked above */
    kk = (double *) malloc(outSize * ksize * sizeof(double));
    if ( ! kk) {
        return 0;
    }

    /* malloc check ok, ksize*sizeof(double) > 2*sizeof(int) */
    bounds = (int *) malloc(outSize * 2 * sizeof(int));
    if ( ! bounds) {
        free(kk);
        return 0;
    }

    for (xx = 0; xx < outSize; xx++) {
        center = in0 + (xx + 0.5) * scale;
        // printf("center = %f\n", center);
        ww = 0.0;
        ss = 1.0 / filterscale;
        // Round the value
        xmin = (int) (center - support + 0.5); // -1.8 -> -2 ,,  3.7 - 5 + 0.5 -> -0.8 . -1
        // printf("support = %f\n", support);
        // printf("xmin = %d\n", xmin);
        // TODO
        // if (xmin < 0)
        //     xmin = 0;
        // Round the value
        xmax = (int) (center + support + 0.5); // 3.2 + 5 + 0.5 -> 8 . 3.7 + 5 + 0.5  -> 9.2  9
        // printf("xmax = %d\n", xmax);
        
        // TODO
        // if (xmax > inSize)
        //     xmax = inSize;
        
        xmax -= xmin;  // 10, 10 0..
        
        k = &kk[xx * ksize];
        for (x = 0; x < xmax; x++) {
            if(x + xmin < 0 || x + xmin >= (int)(in1 - in0) ){
                k[x] = 0;
                continue;
            }
            
            double w = filterp->filter((x + xmin - center + 0.5) * ss);
               
            k[x] = w;
            ww += w;
        }
        for (x = 0; x < xmax; x++) {
            if (ww != 0.0)
                k[x] /= ww;
        }
        // Remaining values should stay empty if they are used despite of xmax.
        for (; x < ksize; x++) {
            k[x] = 0;
        }
        // printf("xmin = %d :: xmax = %d\n", xmin, xmax);
        // for (x = 0; x < xmax; x++)
        //     printf("%f ", k[x]);
        // printf("\n");
        bounds[xx * 2 + 0] = xmin;
        bounds[xx * 2 + 1] = xmax;
    }
    *boundsp = bounds;
    *kkp = kk;
    return ksize;
}


void
normalize_coeffs_8bpc(int outSize, int ksize, double *prekk)
{
    int x;
    INT32 *kk;

    // use the same buffer for normalized coefficients
    kk = (INT32 *) prekk;

    for (x = 0; x < outSize * ksize; x++) {
        if (prekk[x] < 0) {

            kk[x] = (int) (-0.5 + prekk[x] * (1 << PRECISION_BITS));
       
        } else {

            kk[x] = (int) (0.5 + prekk[x] * (1 << PRECISION_BITS));
        }
    }
}
__global__ void verticalKernel( unsigned char *pOut, unsigned char *pIn, 
                                int *bounds,  int* kk, 
                                int dst_width, int dst_height, 
                                int ksize, int inpStride, int outStride,
                                int inpHt,
                                int channels){        
    int dx = threadIdx.x;
    int dy = threadIdx.y;
    int ss0;

    int *k = &kk[dy * ksize];
    int ymin = bounds[dy * 2 + 0];
    int ymax = bounds[dy * 2 + 1];
    for (int c = 0; c < channels; c++){ // channel - 1
        ss0 = 1 << (PRECISION_BITS -1);
        for (int y = 0; y < ymax; y++)
        {   
            if(y + ymin >= 0 && y + ymin < inpHt){
                // if(xx == 12)
                    // printf("%d ", ((UINT8) pIn[(y + ymin)*inpStride + channels*xx + c]) * k[y]);
                ss0 += ((UINT8) pIn[(y + ymin)*inpStride + channels*dx + c]) * k[y];
            }
        }
        // if(xx == 12)
            // printf("\n");                  
        pOut[dy*outStride + channels*dx + c] = clip8_cuda(ss0);
    }
    
}


void
ImagingResampleVertical_8bpc(unsigned char *pOut, unsigned char *pIn, int offset,
                             int ksize, int *bounds, 
                             double *prekk, int inpWd, int inpHt, int inpStride, 
                             int outWd, int outHt, int outStride, 
                             int imType, int channels)
{
    int ss0, ss1, ss2, ss3;
    int xx, yy, y, ymin, ymax;
    int *k, *kk;
    int c;

    // use the same buffer for normalized coefficients
    kk = (INT32 *) prekk;
    normalize_coeffs_8bpc(outHt, ksize, prekk);
    
    printf("\n");
    printf("CALL VERTICAL \n");
    printf("%d %d\n", outHt, outWd);
    printf("input Height : %d \n", inpHt);
    printf("ksize : %d \n", ksize);
    printf("outStride : %d \n", outStride);
    printf("inpStride : %d \n", inpStride);
    
    dim3 dimBlock(outWd,outHt);
    
    int outSize = 32;
    int size = 32;
    int *bounds_gpu;
    int *kk_gpu;

    unsigned char *pOut_gpu;
    unsigned char *pIn_gpu;

    hipMalloc((void**)&bounds_gpu, outSize * 2 * sizeof(int));
    hipMalloc((void**)&kk_gpu, outSize * ksize * sizeof(float));
    
    hipMalloc((void**)&pIn_gpu, 32 * inpHt * 1 * 4  );

    hipMemcpy( bounds_gpu , bounds ,outSize * 2 * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy( kk_gpu , kk ,outSize * ksize * sizeof(int),hipMemcpyHostToDevice);

    // TODO xsize is 32
    hipMemcpy(pIn_gpu, pIn ,32 * inpHt * 1 * 4 ,hipMemcpyHostToDevice);

    // Pout init
    hipMalloc((void**)&pOut_gpu, size * size );
    
    printf("CALL VERTICAL cuda\n");
    verticalKernel<<<1, dimBlock>>>(pOut_gpu, pIn_gpu, bounds_gpu, kk_gpu, outWd, outHt, ksize, inpStride, outStride, inpHt, channels);
    hipDeviceSynchronize();
    hipMemcpy(pOut,pOut_gpu,size*size ,hipMemcpyDeviceToHost);
    for (yy = 0; yy < outHt; yy++) {
        for (xx = 0; xx < outWd; xx++) {
            for (c = 0; c < channels; c++){ // channel - 1
                    printf("%d ", (pOut[yy*outStride + channels*xx + c]));
                }
            }
    }

    printf("\n");    

    // CPU call    
    // for (yy = 0; yy < outHt; yy++) {
    //     k = &kk[yy * ksize];
    //     ymin = bounds[yy * 2 + 0];
    //     ymax = bounds[yy * 2 + 1];
    //     for (xx = 0; xx < outWd; xx++) {
    //         for (c = 0; c < channels; c++){ // channel - 1
    //                 ss0 = 1 << (PRECISION_BITS -1);
    //                 for (y = 0; y < ymax; y++)
    //                 {   
    //                     if(y + ymin >= 0 && y + ymin < inpHt){
    //                         // if(xx == 12)
    //                             // printf("%d ", ((UINT8) pIn[(y + ymin)*inpStride + channels*xx + c]) * k[y]);
    //                         ss0 += ((UINT8) pIn[(y + ymin)*inpStride + channels*xx + c]) * k[y];
    //                     }
    //                 }
    //                 // if(xx == 12)
    //                     // printf("\n");                  
    //                 pOut[yy*outStride + channels*xx + c] = clip8(ss0);
    //             }
    //         }
    // }
    
    // printf("\n");    
    // for (yy = 0; yy < outHt; yy++) {
    //     k = &kk[yy * ksize];
    //     ymin = bounds[yy * 2 + 0];
    //     ymax = bounds[yy * 2 + 1];
    //     for (xx = 0; xx < outWd; xx++) {
    //         for (c = 0; c < channels; c++){ // channel - 1
    //                 printf("%d ", (pOut[yy*outStride + channels*xx + c]));
    //             }
    //         }
    // }
    
}


__global__ void horizontalKernel(unsigned char *pOut, unsigned char *pIn, 
                                int *bounds,  int* kk, 
                                int dst_width, int dst_height, 
                                int ksize, 
                                int inpStride, int outStride,
                                int inpWd,
                                int channels){
    
    int dx = threadIdx.x;
    int dy = threadIdx.y;
    int ss0;

    
    int xmin = bounds[dx * 2 + 0];
    int xmax = bounds[dx * 2 + 1];
    int *k = &kk[dx * ksize];
    for (int c = 0; c < channels; c++){
        ss0 = 1 << (PRECISION_BITS -1);
        for (int x = 0; x < xmax ; x++)
        {
            if(x + xmin >= 0 && x + xmin < inpWd){
                ss0 += ((UINT8) pIn[inpStride*(dy + 0) + channels*(x + xmin) + c]) * k[x];
            }
        }
        pOut[dy*outStride + channels*dx + c] = clip8_cuda(ss0);
    }

} 

void
ImagingResampleHorizontal_8bpc(unsigned char *pOut, unsigned char *pIn, int offset,
                               int ksize, int *bounds, double *prekk, int inpWd, int inpHt, int inpStride, int outWd, int outHt, int outStride, int imType, int channels)
{
    int ss0, ss1, ss2, ss3;
    int xx, yy, x, xmin, xmax;
    int *k, *kk;
    int c;
    kk = (int *) prekk;
    normalize_coeffs_8bpc(outWd, ksize, prekk);
    
    printf("\n");
    printf("CALL HORIZONTAL \n");
    printf("%d %d\n", outHt, outWd);
    printf("input Width : %d \n", inpWd);
    printf("input Height : %d \n", inpHt);
    
    printf("ksize : %d \n", ksize);
    printf("outStride : %d \n", outStride);
    printf("inpStride : %d \n", inpStride);
    
    dim3 dimBlock(outWd,outHt);
    
    int outSize = 32;
    int xsize = 32;
    int *bounds_gpu;
    int *kk_gpu;

    unsigned char *pOut_gpu;
    unsigned char *pIn_gpu;

    hipMalloc((void**)&bounds_gpu, outSize * 2 * sizeof(int));
    hipMalloc((void**)&kk_gpu, outSize * ksize * sizeof(float));
    
    hipMalloc((void**)&pIn_gpu, inpWd * inpHt * 1);

    hipMemcpy( bounds_gpu , bounds ,outSize * 2 * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy( kk_gpu , kk ,outSize * ksize * sizeof(int),hipMemcpyHostToDevice);

    
    hipMemcpy(pIn_gpu, pIn , inpWd * inpHt * 1 ,hipMemcpyHostToDevice);

    // TODO xsize is 32 
    hipMalloc((void**)&pOut_gpu,  32 * inpHt * 1 * 4 );
    
    // printf("CALL HORIZONTAL cuda\n");
    // horizontalKernel<<<1, dimBlock>>>(pOut_gpu, pIn_gpu, bounds_gpu, kk_gpu, outWd, outHt, ksize, inpStride, outStride, inpWd, channels);
    // cudaDeviceSynchronize();
    // cudaMemcpy(pOut,pOut_gpu, 32 * inpHt * 1 * 4 , cudaMemcpyDeviceToHost);


    for (yy = 0; yy < outHt; yy++) {
        for (xx = 0; xx < outWd; xx++) {
            xmin = bounds[xx * 2 + 0];
            xmax = bounds[xx * 2 + 1];
            k = &kk[xx * ksize];
            for (c = 0; c < channels; c++){
                ss0 = 1 << (PRECISION_BITS -1);
                for (x = 0; x < xmax ; x++)
                {
                    if(x + xmin >= 0 && x + xmin < inpWd){
                        ss0 += ((UINT8) pIn[inpStride*(yy + 0) + channels*(x + xmin) + c]) * k[x];
                    }
                }
                pOut[yy*outStride + channels*xx + c] = clip8(ss0);
            }
        }
    }
    for (yy = 0; yy < outHt; yy++) {
        for (xx = 0; xx < outWd; xx++) {
            for (c = 0; c < channels; c++){
                printf("%d ", pOut[yy*outStride + channels*xx + c]);
            }
        }
    }


}

int ImagingResampleInner(   unsigned char *pIn, unsigned char *pOut,
                            int inpWd, int inpHt, int inpStride, int xsize, int ysize, int outStride,
                            struct filter *filterp, float box[4],
                     ResampleFunction ResampleHorizontal,
                     ResampleFunction ResampleVertical, 
                     int imType, int channels)
{
    unsigned char *pImTemp = NULL;

    int i, need_horizontal, need_vertical;
    int ybox_first, ybox_last;
    int ksize_horiz, ksize_vert;
    int *bounds_horiz, *bounds_vert;
    double *kk_horiz, *kk_vert;

    need_horizontal = xsize != inpWd || box[0] || box[2] != xsize;
    need_vertical = ysize != inpHt || box[1] || box[3] != ysize;
    

    printf("xsize: %d \n", xsize);
    printf("inpWd: %d \n", inpWd);
    printf("Need Horizontal: %d \n", need_horizontal);
    printf("Need Vertical: %d \n", need_vertical);

    
    ksize_horiz = precompute_coeffs(inpWd, box[0], box[2], xsize,
                                    filterp, &bounds_horiz, &kk_horiz);
    if ( ! ksize_horiz) {
        return -1;
    }

    ksize_vert = precompute_coeffs(inpHt, box[1], box[3], ysize,
                                   filterp, &bounds_vert, &kk_vert);

    if ( ! ksize_vert) {
        free(bounds_horiz);
        free(kk_horiz);
        free(bounds_vert);
        free(kk_vert);
        return -1;
    }

    // First used row in the source image
    ybox_first = bounds_vert[0];
    // Last used row in the source image
    ybox_last = bounds_vert[ysize*2 - 2] + bounds_vert[ysize*2 - 1];


    /* two-pass resize, horizontal pass */
    if (1) {
        int stride;
        // Shift bounds for vertical pass
        // TODO
        // for (i = 0; i < ysize; i++) {
        //     bounds_vert[i * 2] -= ybox_first;
        // }
    if (1)
        pImTemp = (unsigned char *)malloc(xsize * inpHt * channels * 4);
    else pImTemp = pOut;
    
    stride = 1?xsize:outStride;

    if (pImTemp) {
        
        ResampleHorizontal(pImTemp, pIn, ybox_first,
                            ksize_horiz, 
                            bounds_horiz, 
                            kk_horiz, 
                            inpWd, inpHt, inpStride, 
                            xsize, inpHt, stride, 
                            imType, channels
                            );
        
        //  ResampleHorizontal<<<1,1>>>(pImTemp, pIn, ybox_first,
        //                     ksize_horiz, 
        //                     bounds_horiz, 
        //                     kk_horiz, 
        //                     inpWd, inpHt, inpStride, 
        //                     xsize, inpHt, stride, 
        //                     imType, channels,
        //                     jobs);
    
    }
    free(bounds_horiz);
    free(kk_horiz);
    if ( ! pImTemp) {
        free(bounds_vert);
        free(kk_vert);
        return -1;
    }
    //imOut = imIn = imTemp;
    } else {
        // Free in any case
        free(bounds_horiz);
        free(kk_horiz);
    }

    /* vertical pass */
    if (1) {
        unsigned char *pIn2;
        int wd;
        int stride = 1?xsize:inpStride;
        pIn2 = 1?pImTemp:pIn;

        if (1) {
            /* imIn can be the original image or horizontally resampled one */
            ResampleVertical(  pOut, pIn2, 0,
                                ksize_vert, 
                                bounds_vert, 
                                kk_vert, 
                                xsize, inpHt, stride, xsize, ysize, 
                                outStride, imType, channels);

            // ResampleVertical<<<1,1>>>(  pOut, pIn2, 0,
            //                     ksize_vert, 
            //                     bounds_vert, 
            //                     kk_vert, 
            //                     xsize, inpHt, stride, xsize, ysize, 
            //                     outStride, imType, channels,
            //                     jobs);

        }

        /* it's safe to call ImagingDelete with empty value
           if previous step was not performed. */
        free(pImTemp);
        free(bounds_vert);
        free(kk_vert);
        return 0;
        /*if ( ! imOut) {
            return NULL;
        }*/
    } else {
        // Free in any case
        free(bounds_vert);
        free(kk_vert);
    }

    /* none of the previous steps are performed, copying */
    if ( ! (need_horizontal || need_vertical)) {
    //printf("memcpy only\n");
        //memcpy(pOut, pIn, xsize*ysize*((imType == IMAGING_TYPE_UINT8)?1:4)*channels);
    int i;
    for (i = 0; i < ysize; i++)
        memcpy(pOut + i*outStride, pIn + i*inpStride, xsize*channels*((imType == IMAGING_TYPE_UINT8)?1:4));
    }

    return 0;
}


int ImagingResample(unsigned char *pIn, unsigned char *pOut, int inpWd, int inpHt, int inpStride, int xsize, int ysize, int outStride, int filter, float box[4], int imType, int channels)
{
    struct filter *filterp;
    // ResampleFunctionCuda ResampleHorizontal;
    // ResampleFunctionCuda ResampleVertical;
    ResampleFunction ResampleHorizontal;
    ResampleFunction ResampleVertical;


    
        switch(imType) {
            case IMAGING_TYPE_UINT8:
                ResampleHorizontal = ImagingResampleHorizontal_8bpc; //ImagingResampleHorizontal_8bpc_kernel;  //ImagingResampleHorizontal_8bpc;
                ResampleVertical = ImagingResampleVertical_8bpc; // ImagingResampleVertical_8bpc_kernel; // ImagingResampleVertical_8bpc;
                break;
            case IMAGING_TYPE_INT32:
            case IMAGING_TYPE_FLOAT32:
                //ResampleHorizontal = ImagingResampleHorizontal_32bpc;
                //ResampleVertical = ImagingResampleVertical_32bpc;
                //break;
            default:
                return -1;
        }

    /* check filter */
    switch (filter) {
    case IMAGING_TRANSFORM_BOX:
        filterp = &BOX;
        break;
    case IMAGING_TRANSFORM_BILINEAR:
        filterp = &BILINEAR;
        break;
    case IMAGING_TRANSFORM_HAMMING:
        filterp = &HAMMING;
        break;
    case IMAGING_TRANSFORM_BICUBIC:
        filterp = &BICUBIC;
        break;
    case IMAGING_TRANSFORM_LANCZOS:
        filterp = &LANCZOS;
        break;
    default:
        return -1;
    }

    return ImagingResampleInner(pIn, pOut, inpWd, inpHt, inpStride, xsize, ysize, outStride, filterp, box,
                                ResampleHorizontal, ResampleVertical, imType, channels);
}


// modified resize routine
int resizeModPIL(unsigned char *pIn, unsigned char *pOut, int inpWd, int inpHt, int inpStride, int outWd, int outHt, int outStride, int channels)
{
    //Imaging imIn;
    //Imaging imOut;

    int xsize, ysize;
    int filter = IMAGING_TRANSFORM_LANCZOS;
    float box[4] = {0, 0, 0, 0};
    int imType = IMAGING_TYPE_UINT8;
    //imIn = self->image;
    box[2] = inpWd;
    box[3] = inpHt;
    
    xsize = outWd;
    ysize = outHt;
    
    
    if (xsize < 1 || ysize < 1) {
        return -1;//ImagingError_ValueError("height and width must be > 0");
    }

    if (box[0] < 0 || box[1] < 0) {
        return -1;//ImagingError_ValueError("box offset can't be negative");
    }

    if (box[2] > inpWd || box[3] > inpHt) {
        return -1;//ImagingError_ValueError("box can't exceed original image size");
    }

    if (box[2] - box[0] < 0 || box[3] - box[1] < 0) {
        return -1;//ImagingError_ValueError("box can't be empty");
    }

    // If box's coordinates are int and box size matches requested size
    if (0)/*(box[0] - (int) box[0] == 0 && box[2] - box[0] == xsize
            && box[1] - (int) box[1] == 0 && box[3] - box[1] == ysize) */{
        //imOut = ImagingCrop(imIn, box[0], box[1], box[2], box[3]);
    }
    else if (filter == IMAGING_TRANSFORM_NEAREST) {
        double a[6];

        memset(a, 0, sizeof a);
        a[0] = (double) (box[2] - box[0]) / xsize;
        a[4] = (double) (box[3] - box[1]) / ysize;
        a[2] = box[0];
        a[5] = box[1];

        /*imOut = ImagingNewDirty(imIn->mode, xsize, ysize);

        imOut = ImagingTransform(
            imOut, imIn, IMAGING_TRANSFORM_AFFINE,
            0, 0, xsize, ysize,
            a, filter, 1);*/
    }
    else {
      printf("calling imagingresample\n");
      return ImagingResample(pIn, pOut, inpWd, inpHt, inpStride, xsize, ysize, outStride, filter, box, imType, channels);
    }

    return 0;
}
int main(int argc, char *argv[])
{
    // car1.jpg JPEG 350x174 350x174+0+0 8-bit sRGB 19.7KB 0.000u 0:00.000
    // car2.jpg[1] JPEG 572x342 572x342+0+0 8-bit sRGB 45.8KB 0.000u 0:00.000
    // car3.jpg[2] JPEG 228x174 228x174+0+0 8-bit sRGB 15.2KB 0.000u 0:00.000

    unsigned char *pIn, *pOut;
    int ret, i;
    int inpWd = 1881;
    int inpHt = 926;
    int inpStride = 1881;
    int outWd = 32;
    int outHt = 32;
    int outStride = 32;
    int nCh = 1;
    FILE *fp1;
    FILE *fp = fopen("./LM_crop.raw", "rb");
    // FILE *fp = fopen("./car1.raw", "rb");
    // FILE *fp = fopen("./car2.raw", "rb");
    // FILE *fp = fopen("./car3.raw", "rb");
    // FILE *fp = fopen("./pixel.raw", "rb");
    
    pIn = (unsigned char *)malloc(inpStride*inpHt*nCh);
    pOut = (unsigned char *)malloc(outStride*outHt*nCh);
    
    for (i = 0; i < inpHt; i++)
        fread(pIn+i*inpStride, 1, inpWd*nCh, fp);
    fclose(fp);

    ret = resizeModPIL(pIn, pOut, inpWd, inpHt, inpStride, outWd, outHt, outStride, nCh);
    printf("return status = %d\n", ret);
    fp1 = fopen("./LM_resized.raw", "wb");
    // fp1 = fopen("./car1_resized.raw", "wb");
    // fp1 = fopen("./car2_resized.raw", "wb");
    // fp1 = fopen("./car3_resized.raw", "wb");
    // fp1 = fopen("./pixel_resized.raw", "wb");
    
    
    for (i = 0; i < outHt; i++)
        fwrite(pOut + i * outStride, 1, outWd*nCh, fp1);
    fclose(fp1);
}
