
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define ROUND_UP(f) ((int) ((f) >= 0.0 ? (f) + 0.5F : (f) - 0.5F))
#define UINT8 unsigned char
#define INT32 int
#define INT_MAX 0x7fffffff

/* pixel types */
#define IMAGING_TYPE_UINT8 0
#define IMAGING_TYPE_INT32 1
#define IMAGING_TYPE_FLOAT32 2
#define IMAGING_TYPE_SPECIAL 3 /* check mode for details */

#define IMAGING_MODE_LENGTH 6+1 /* Band names ("1", "L", "P", "RGB", "RGBA", "CMYK", "YCbCr", "BGR;xy") */


/* standard transforms */
#define IMAGING_TRANSFORM_AFFINE 0
#define IMAGING_TRANSFORM_PERSPECTIVE 2
#define IMAGING_TRANSFORM_QUAD 3


/* standard filters */
#define IMAGING_TRANSFORM_NEAREST 0
#define IMAGING_TRANSFORM_BOX 4
#define IMAGING_TRANSFORM_BILINEAR 2
#define IMAGING_TRANSFORM_HAMMING 5
#define IMAGING_TRANSFORM_BICUBIC 3
#define IMAGING_TRANSFORM_LANCZOS 1

typedef void (*ResampleFunction)(unsigned char *pOut, unsigned char *pIn, int offset,
                               int ksize, int *bounds, double *prekk, int inpWd, int inpHt, int inpStride, int outWd, int outHt, int outStride, int imType, int channels);
struct filter {
    double (*filter)(double x);
    double support;
};

static inline double box_filter(double x)
{
    if (x >= -0.5 && x < 0.5)
        return 1.0;
    return 0.0;
}

static inline double bilinear_filter(double x)
{
    if (x < 0.0)
        x = -x;
    if (x < 1.0)
        return 1.0-x;
    return 0.0;
}

static inline double hamming_filter(double x)
{
    if (x < 0.0)
        x = -x;
    if (x == 0.0)
        return 1.0;
    if (x >= 1.0)
        return 0.0;
    x = x * M_PI;
    return sin(x) / x * (0.54f + 0.46f * cos(x));
}

static inline double bicubic_filter(double x)
{
    /* https://en.wikipedia.org/wiki/Bicubic_interpolation#Bicubic_convolution_algorithm */
#define a -0.5
    if (x < 0.0)
        x = -x;
    if (x < 1.0)
        return ((a + 2.0) * x - (a + 3.0)) * x*x + 1;
    if (x < 2.0)
        return (((x - 5) * x + 8) * x - 4) * a;
    return 0.0;
#undef a
}

static inline double sinc_filter(double x)
{
    if (x == 0.0)
        return 1.0;
    x = x * M_PI;
    return sin(x) / x;
}

static inline double lanczos_filter(double x)
{
    /* truncated sinc */
    if (-3.0 <= x && x < 3.0)
        return sinc_filter(x) * sinc_filter(x/3);
    return 0.0;
}

static struct filter BOX = { box_filter, 0.5 };
static struct filter BILINEAR = { bilinear_filter, 1.0 };
static struct filter HAMMING = { hamming_filter, 1.0 };
static struct filter BICUBIC = { bicubic_filter, 2.0 };
static struct filter LANCZOS = { lanczos_filter, 3.0 };


/* 8 bits for result. Filter can have negative areas.
   In one cases the sum of the coefficients will be negative,
   in the other it will be more than 1.0. That is why we need
   two extra bits for overflow and int type. */
#define PRECISION_BITS (32 - 8 - 2)


/* Handles values form -640 to 639. */
UINT8 _clip8_lookups[1280] = {
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15,
    16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31,
    32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47,
    48, 49, 50, 51, 52, 53, 54, 55, 56, 57, 58, 59, 60, 61, 62, 63,
    64, 65, 66, 67, 68, 69, 70, 71, 72, 73, 74, 75, 76, 77, 78, 79,
    80, 81, 82, 83, 84, 85, 86, 87, 88, 89, 90, 91, 92, 93, 94, 95,
    96, 97, 98, 99, 100, 101, 102, 103, 104, 105, 106, 107, 108, 109, 110, 111,
    112, 113, 114, 115, 116, 117, 118, 119, 120, 121, 122, 123, 124, 125, 126, 127,
    128, 129, 130, 131, 132, 133, 134, 135, 136, 137, 138, 139, 140, 141, 142, 143,
    144, 145, 146, 147, 148, 149, 150, 151, 152, 153, 154, 155, 156, 157, 158, 159,
    160, 161, 162, 163, 164, 165, 166, 167, 168, 169, 170, 171, 172, 173, 174, 175,
    176, 177, 178, 179, 180, 181, 182, 183, 184, 185, 186, 187, 188, 189, 190, 191,
    192, 193, 194, 195, 196, 197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207,
    208, 209, 210, 211, 212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223,
    224, 225, 226, 227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239,
    240, 241, 242, 243, 244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
    255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
};

UINT8 *clip8_lookups = &_clip8_lookups[640];

static inline UINT8 clip8(int in)
{
    //printf("%d\n", in);
    return clip8_lookups[in >> PRECISION_BITS];
}


int
precompute_coeffs(int inSize, float in0, float in1, int outSize,
                  struct filter *filterp, int **boundsp, double **kkp) {
    double support, scale, filterscale;
    double center, ww, ss;
    int xx, x, ksize, xmin, xmax;
    int *bounds;
    double *kk, *k;

    /* prepare for horizontal stretch */
    //printf("outsize = %d :: in1 = %f :: in0 = %f \n", outSize, in1, in0);
    filterscale = scale = (double) (in1 - in0) / outSize;
    if (filterscale < 1.0) {
        filterscale = 1.0;
    }

    /* determine support size (length of resampling filter) */
    support = filterp->support * filterscale;

    /* maximum number of coeffs */
    ksize = (int) ceil(support) * 2 + 1;

    printf("ksize = %d\n", ksize);
    printf("support = %f\n", ceil(support));
    printf("filterscale = %f\n", filterscale);
    
    // check for overflow
    if (outSize > INT_MAX / (ksize * sizeof(double))) {
        return 0;
    }

    /* coefficient buffer */
    /* malloc check ok, overflow checked above */
    kk = (double *) malloc(outSize * ksize * sizeof(double));
    if ( ! kk) {
        return 0;
    }

    /* malloc check ok, ksize*sizeof(double) > 2*sizeof(int) */
    bounds = (int *) malloc(outSize * 2 * sizeof(int));
    if ( ! bounds) {
        free(kk);
        return 0;
    }

    for (xx = 0; xx < outSize; xx++) {
        center = in0 + (xx + 0.5) * scale;
        ww = 0.0;
        ss = 1.0 / filterscale;
        // Round the value
        xmin = (int) (center - support + 0.5);
        if (xmin < 0)
            xmin = 0;
        printf("support = %f\n", support);
        printf("xmin = %d\n", xmin);
        
        // Round the value
        xmax = (int) (center + support + 0.5);
        if (xmax > inSize)
            xmax = inSize;
        
        printf("xmax = %d\n", xmax);

        xmax -= xmin;
        k = &kk[xx * ksize];
        for (x = 0; x < xmax; x++) {
            double w = filterp->filter((x + xmin - center + 0.5) * ss);
            k[x] = w;
            ww += w;
        }
        for (x = 0; x < xmax; x++) {
            if (ww != 0.0)
                k[x] /= ww;
        }
	
        // Remaining values should stay empty if they are used despite of xmax.
        for (; x < ksize; x++) {
            k[x] = 0;
        }
        printf("xmin = %d :: xmax = %d\n", xmin, xmax);
        for (x = 0; x < xmax; x++)
            printf("%f ", k[x]);
        printf("\n");
        bounds[xx * 2 + 0] = xmin;
        bounds[xx * 2 + 1] = xmax;
    }
    *boundsp = bounds;
    *kkp = kk;
    return ksize;
}


void
normalize_coeffs_8bpc(int outSize, int ksize, double *prekk)
{
    int x;
    INT32 *kk;

    // use the same buffer for normalized coefficients
    kk = (INT32 *) prekk;

    for (x = 0; x < outSize * ksize; x++) {
        if (prekk[x] < 0) {
            kk[x] = (int) (-0.5 + prekk[x] * (1 << PRECISION_BITS));
        } else {
            kk[x] = (int) (0.5 + prekk[x] * (1 << PRECISION_BITS));
        }
    }
}


void
ImagingResampleVertical_8bpc(unsigned char *pOut, unsigned char *pIn, int offset,
                             int ksize, int *bounds, double *prekk, int inpWd, int inpHt, int inpStride, int outWd, int outHt, int outStride, int imType, int channels)
{
    int ss0, ss1, ss2, ss3;
    int xx, yy, y, ymin, ymax;
    int *k, *kk;
    int c;

    // use the same buffer for normalized coefficients
    kk = (INT32 *) prekk;
    normalize_coeffs_8bpc(outHt, ksize, prekk);
    printf("calling vertical resample\n");
    for (yy = 0; yy < outHt; yy++) {
        k = &kk[yy * ksize];
        ymin = bounds[yy * 2 + 0];
        ymax = bounds[yy * 2 + 1];
        for (xx = 0; xx < outWd; xx++) {
	    for (c = 0; c < channels; c++){
                ss0 = 1 << (PRECISION_BITS -1);
                for (y = 0; y < ymax; y++){
                    if(xx == 12)
                        printf("%d ", ((UINT8) pIn[(y + ymin)*inpStride + channels*xx + c]) * k[y]);
                    ss0 += ((UINT8) pIn[(y + ymin)*inpStride + channels*xx + c]) * k[y];
                
                }
                if(xx == 12)
                    printf("\n");
                
                pOut[yy*outStride + channels*xx + c] = clip8(ss0);
            }
        }
    }
}


void
ImagingResampleHorizontal_8bpc(unsigned char *pOut, unsigned char *pIn, int offset,
                               int ksize, int *bounds, double *prekk, int inpWd, int inpHt, int inpStride, int outWd, int outHt, int outStride, int imType, int channels)
{
    int ss0, ss1, ss2, ss3;
    int xx, yy, x, xmin, xmax;
    int *k, *kk;
    int c;
    printf("offset = %d\n", offset);
    // use the same buffer for normalized coefficients
    kk = (int *) prekk;
    normalize_coeffs_8bpc(outWd, ksize, prekk);
	//printf("calling horizontal resample\n");
    for (yy = 0; yy < outHt; yy++) {
        for (xx = 0; xx < outWd; xx++) {
            xmin = bounds[xx * 2 + 0];
            xmax = bounds[xx * 2 + 1];
            k = &kk[xx * ksize];
            for (c = 0; c < channels; c++){

                ss0 = 1 << (PRECISION_BITS -1);
              for (x = 0; x < xmax; x++)
                {   
                    if(yy == 12){
                        printf( "%d " ,((UINT8) pIn[inpStride*(yy + offset) + channels*(x + xmin) + c]) * k[x]);
                    }
                    ss0 += ((UINT8) pIn[inpStride*(yy + offset) + channels*(x + xmin) + c]) * k[x];
                }
                if(yy == 12){
                    printf("\n");
                }
                pOut[yy*outStride + channels*xx + c] = clip8(ss0);
		//printf("%d\n", clip8(ss0));
            }
        }
    } 
}

int ImagingResampleInner(unsigned char *pIn, unsigned char *pOut, int inpWd, int inpHt, int inpStride, int xsize, int ysize, int outStride,
                     struct filter *filterp, float box[4],
                     ResampleFunction ResampleHorizontal,
                     ResampleFunction ResampleVertical, int imType, int channels)
{
    unsigned char *pImTemp = NULL;

    int i, need_horizontal, need_vertical;
    int ybox_first, ybox_last;
    int ksize_horiz, ksize_vert;
    int *bounds_horiz, *bounds_vert;
    double *kk_horiz, *kk_vert;

    need_horizontal = xsize != inpWd || box[0] || box[2] != xsize;
    need_vertical = ysize != inpHt || box[1] || box[3] != ysize;
    
    printf("xsize: %d \n", xsize);
    printf("inpWd: %d \n", inpWd);
    printf("Need Horizontal: %d \n", need_horizontal);
    printf("Need Vertical: %d \n", need_vertical);


    ksize_horiz = precompute_coeffs(inpWd, box[0], box[2], xsize,
                                    filterp, &bounds_horiz, &kk_horiz);
    if ( ! ksize_horiz) {
        return -1;
    }

    ksize_vert = precompute_coeffs(inpHt, box[1], box[3], ysize,
                                   filterp, &bounds_vert, &kk_vert);

    if ( ! ksize_vert) {
        free(bounds_horiz);
        free(kk_horiz);
        free(bounds_vert);
        free(kk_vert);
        return -1;
    }

    // First used row in the source image
    ybox_first = bounds_vert[0];
    // Last used row in the source image
    ybox_last = bounds_vert[ysize*2 - 2] + bounds_vert[ysize*2 - 1];


    /* two-pass resize, horizontal pass */
    if (need_horizontal) {
        int stride;
        // Shift bounds for vertical pass
        for (i = 0; i < ysize; i++) {
            bounds_vert[i * 2] -= ybox_first;
        }
    if (need_vertical)
        pImTemp = (unsigned char *)malloc(xsize * inpHt * channels * 4);
    else pImTemp = pOut;
    
    stride = need_vertical?xsize:outStride;
        if (pImTemp) {
            ResampleHorizontal(pImTemp, pIn, ybox_first,
                               ksize_horiz, bounds_horiz, kk_horiz, inpWd, inpHt, inpStride, xsize, inpHt, stride, imType, channels);
        }
        free(bounds_horiz);
        free(kk_horiz);
        if ( ! pImTemp) {
            free(bounds_vert);
            free(kk_vert);
            return -1;
        }
        //imOut = imIn = imTemp;
    } else {
        // Free in any case
        free(bounds_horiz);
        free(kk_horiz);
    }

    /* vertical pass */
    if (need_vertical) {
        unsigned char *pIn2;
        int wd;
        int stride = need_horizontal?xsize:inpStride;
        pIn2 = need_horizontal?pImTemp:pIn;
        if (1) {
            /* imIn can be the original image or horizontally resampled one */
            ResampleVertical(pOut, pIn2, 0,
                             ksize_vert, bounds_vert, kk_vert, xsize, inpHt, stride, xsize, ysize, outStride, imType, channels);
        }

        /* it's safe to call ImagingDelete with empty value
           if previous step was not performed. */
        free(pImTemp);
        free(bounds_vert);
        free(kk_vert);
        return 0;
        /*if ( ! imOut) {
            return NULL;
        }*/
    } else {
        // Free in any case
        free(bounds_vert);
        free(kk_vert);
    }

    /* none of the previous steps are performed, copying */
    if ( ! (need_horizontal || need_vertical)) {
    //printf("memcpy only\n");
        //memcpy(pOut, pIn, xsize*ysize*((imType == IMAGING_TYPE_UINT8)?1:4)*channels);
    int i;
    for (i = 0; i < ysize; i++)
        memcpy(pOut + i*outStride, pIn + i*inpStride, xsize*channels*((imType == IMAGING_TYPE_UINT8)?1:4));
    }

    return 0;
}


int ImagingResample(unsigned char *pIn, unsigned char *pOut, int inpWd, int inpHt, int inpStride, int xsize, int ysize, int outStride, int filter, float box[4], int imType, int channels)
{
    struct filter *filterp;
    ResampleFunction ResampleHorizontal;
    ResampleFunction ResampleVertical;


    
        switch(imType) {
            case IMAGING_TYPE_UINT8:
                ResampleHorizontal = ImagingResampleHorizontal_8bpc;
                ResampleVertical = ImagingResampleVertical_8bpc;
                break;
            case IMAGING_TYPE_INT32:
            case IMAGING_TYPE_FLOAT32:
                //ResampleHorizontal = ImagingResampleHorizontal_32bpc;
                //ResampleVertical = ImagingResampleVertical_32bpc;
                //break;
            default:
                return -1;
        }

    /* check filter */
    switch (filter) {
    case IMAGING_TRANSFORM_BOX:
        filterp = &BOX;
        break;
    case IMAGING_TRANSFORM_BILINEAR:
        filterp = &BILINEAR;
        break;
    case IMAGING_TRANSFORM_HAMMING:
        filterp = &HAMMING;
        break;
    case IMAGING_TRANSFORM_BICUBIC:
        filterp = &BICUBIC;
        break;
    case IMAGING_TRANSFORM_LANCZOS:
        filterp = &LANCZOS;
        break;
    default:
        return -1;
    }

    return ImagingResampleInner(pIn, pOut, inpWd, inpHt, inpStride, xsize, ysize, outStride, filterp, box,
                                ResampleHorizontal, ResampleVertical, imType, channels);
}


// modified resize routine
int resizeModPIL(unsigned char *pIn, unsigned char *pOut, int inpWd, int inpHt, int inpStride, int outWd, int outHt, int outStride, int channels)
{
    //Imaging imIn;
    //Imaging imOut;

    int xsize, ysize;
    int filter = IMAGING_TRANSFORM_LANCZOS;
    float box[4] = {0, 0, 0, 0};
    int imType = IMAGING_TYPE_UINT8;
    //imIn = self->image;
    box[2] = inpWd;
    box[3] = inpHt;
    
    xsize = outWd;
    ysize = outHt;
    
    
    if (xsize < 1 || ysize < 1) {
        return -1;//ImagingError_ValueError("height and width must be > 0");
    }

    if (box[0] < 0 || box[1] < 0) {
        return -1;//ImagingError_ValueError("box offset can't be negative");
    }

    if (box[2] > inpWd || box[3] > inpHt) {
        return -1;//ImagingError_ValueError("box can't exceed original image size");
    }

    if (box[2] - box[0] < 0 || box[3] - box[1] < 0) {
        return -1;//ImagingError_ValueError("box can't be empty");
    }

    // If box's coordinates are int and box size matches requested size
    if (0)/*(box[0] - (int) box[0] == 0 && box[2] - box[0] == xsize
            && box[1] - (int) box[1] == 0 && box[3] - box[1] == ysize) */{
        //imOut = ImagingCrop(imIn, box[0], box[1], box[2], box[3]);
    }
    else if (filter == IMAGING_TRANSFORM_NEAREST) {
        double a[6];

        memset(a, 0, sizeof a);
        a[0] = (double) (box[2] - box[0]) / xsize;
        a[4] = (double) (box[3] - box[1]) / ysize;
        a[2] = box[0];
        a[5] = box[1];

        /*imOut = ImagingNewDirty(imIn->mode, xsize, ysize);

        imOut = ImagingTransform(
            imOut, imIn, IMAGING_TRANSFORM_AFFINE,
            0, 0, xsize, ysize,
            a, filter, 1);*/
    }
    else {
	//printf("calling imagingresample\n");
        return ImagingResample(pIn, pOut, inpWd, inpHt, inpStride, xsize, ysize, outStride, filter, box, imType, channels);
    }

    return 0;
}
int main(int argc, char *argv[])
{

    // car1.jpg JPEG 350x174 350x174+0+0 8-bit sRGB 19.7KB 0.000u 0:00.000
    // car2.jpg[1] JPEG 572x342 572x342+0+0 8-bit sRGB 45.8KB 0.000u 0:00.000
    // car3.jpg[2] JPEG 228x174 228x174+0+0 8-bit sRGB 15.2KB 0.000u 0:00.000

    unsigned char *pIn, *pOut;
    int ret, i;
    int inpWd = 1881;
    int inpHt = 926;
    int inpStride = 1881;
    int outWd = 32;
    int outHt = 32;
    int outStride = 32;
    int nCh = 1;
    FILE *fp1;
    FILE *fp = fopen("./LM_crop.raw", "rb");
    // FILE *fp = fopen("./car1.raw", "rb");
    // FILE *fp = fopen("./car2.raw", "rb");
    // FILE *fp = fopen("./car3.raw", "rb");
    // FILE *fp = fopen("./pixel.raw", "rb");

    pIn = (unsigned char *)malloc(inpStride*inpHt*nCh);
    pOut = (unsigned char *)malloc(outStride*outHt*nCh);
   
    for (i = 0; i < inpHt; i++)
        fread(pIn+i*inpStride, 1, inpWd*nCh, fp);
    fclose(fp);

    ret = resizeModPIL(pIn, pOut, inpWd, inpHt, inpStride, outWd, outHt, outStride, nCh);
    printf("return status = %d\n", ret);
    fp1 = fopen("./LM_resize_original.raw", "wb");
    // fp1 = fopen("./car1_resized_original.raw", "wb");
    // fp1 = fopen("./car2_resized_original.raw", "wb");
    // fp1 = fopen("./car3_resized_original.raw", "wb");
    // fp1 = fopen("./pixel_resized_original.raw", "wb");
    
    for (i = 0; i < outHt; i++)
        fwrite(pOut + i * outStride, 1, outWd*nCh, fp1);
    fclose(fp1);
}